#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include "nvshmem.h"
#include "nvshmemx.h"

#undef CUDA_CHECK
#define CUDA_CHECK(stmt)                                                          \
    do {                                                                          \
        hipError_t result = (stmt);                                              \
        if (hipSuccess != result) {                                              \
            fprintf(stderr, "[%s:%d] cuda failed with %s \n", __FILE__, __LINE__, \
                    hipGetErrorString(result));                                  \
            exit(-1);                                                             \
        }                                                                         \
    } while (0)

#define THREADS_PER_BLOCK 1

__global__ void benchmark_signal_kernel(uint64_t* signal_buf, int mype, int npes, int rounds, uint64_t start_value) {
    int peer = (mype + 1) % npes;
    if (threadIdx.x == 0) {
        for (int i = 0; i < rounds; ++i) {
            uint64_t expected = start_value + i * 2 + mype;
            nvshmem_signal_wait_until(signal_buf, NVSHMEM_CMP_EQ, expected);
            nvshmemx_signal_op(signal_buf, uint64_t(i * 2 + mype + 1), NVSHMEM_SIGNAL_SET, peer);
        }
    }
}

int main(int c, char *v[]) {
    int mype, npes, mype_node;
    uint64_t *signal_buf;

    nvshmem_init();

    mype = nvshmem_my_pe();
    npes = nvshmem_n_pes();
    mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);

    // application picks the device each PE will use
    CUDA_CHECK(hipSetDevice(mype_node));
    signal_buf = (uint64_t *)nvshmem_malloc(sizeof(uint64_t));
    assert(signal_buf != NULL);

    int rounds = 10;
    uint64_t start_value = 0;

    hipEvent_t start, stop;
    float elapsed_ms = 0.0f;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    CUDA_CHECK(hipEventRecord(start, 0));
    // ------------------------------

    benchmark_signal_kernel<<<1, THREADS_PER_BLOCK>>>(signal_buf, mype, npes, rounds, start_value);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // ------------------------------
    CUDA_CHECK(hipEventRecord(stop, 0));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&elapsed_ms, start, stop));
    printf("GPU%d: %.3f ms total, %.3f us/send\n", mype, elapsed_ms, 1000.0f * elapsed_ms / (rounds * 2));
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    // ------------------------------

    nvshmem_free(signal_buf);
    nvshmem_finalize();

    return 0;
}